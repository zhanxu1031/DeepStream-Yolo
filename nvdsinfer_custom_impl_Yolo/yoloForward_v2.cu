/*
 * Created by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

inline __device__ float sigmoidGPU(const float& x) { return 1.0f / (1.0f + __expf(-x)); }

__device__ void softmaxGPU(const float* input, const int bbindex, const int numGridCells, uint z_id,
    const uint numOutputClasses, float temp, float* output)
{
  int i;
  float sum = 0;
  float largest = -INFINITY;
  for (i = 0; i < numOutputClasses; ++i) {
    int val = input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))];
    largest = (val>largest) ? val : largest;
  }
  for (i = 0; i < numOutputClasses; ++i) {
    float e = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] / temp - largest / temp);
    sum += e;
    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] = e;
  }
  for (i = 0; i < numOutputClasses; ++i) {
    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] /= sum;
  }
}

__global__ void gpuRegionLayer(const float* input, float* softmax, float* output, const uint netWidth,
    const uint netHeight, const uint gridSizeX, const uint gridSizeY, const uint numOutputClasses, const uint numBBoxes,
    const uint64_t lastInputSize, const float* anchors)
{
  uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
  uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
  uint z_id = blockIdx.z * blockDim.z + threadIdx.z;

  if (x_id >= gridSizeX || y_id >= gridSizeY || z_id >= numBBoxes) {
    return;
  }

  const int numGridCells = gridSizeX * gridSizeY;
  const int bbindex = y_id * gridSizeX + x_id;

  float xc = (sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)]) + x_id) * netWidth /
      gridSizeX;

  float yc = (sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)]) + y_id) * netHeight /
      gridSizeY;

  float w = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)]) * anchors[z_id * 2] * netWidth /
      gridSizeX;

  float h = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)]) * anchors[z_id * 2 + 1] *
      netHeight / gridSizeY;

  const float objectness = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)]);

  softmaxGPU(input, bbindex, numGridCells, z_id, numOutputClasses, 1.0, softmax);

  float maxProb = 0.0f;
  int maxIndex = -1;

  for (uint i = 0; i < numOutputClasses; ++i) {
    float prob = softmax[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))];
    if (prob > maxProb) {
      maxProb = prob;
      maxIndex = i;
    }
  }

  int count = numGridCells * z_id + bbindex + lastInputSize;

  output[count * 6 + 0] = xc - w * 0.5;
  output[count * 6 + 1] = yc - h * 0.5;
  output[count * 6 + 2] = xc + w * 0.5;
  output[count * 6 + 3] = yc + h * 0.5;
  output[count * 6 + 4] = maxProb * objectness;
  output[count * 6 + 5] = (float) maxIndex;
}

hipError_t cudaRegionLayer(const void* input, void* softmax, void* output, const uint& batchSize,
    const uint64_t& inputSize, const uint64_t& outputSize, const uint64_t& lastInputSize, const uint& netWidth,
    const uint& netHeight, const uint& gridSizeX, const uint& gridSizeY, const uint& numOutputClasses,
    const uint& numBBoxes, const void* anchors, hipStream_t stream);

hipError_t cudaRegionLayer(const void* input, void* softmax, void* output, const uint& batchSize,
    const uint64_t& inputSize, const uint64_t& outputSize, const uint64_t& lastInputSize, const uint& netWidth,
    const uint& netHeight, const uint& gridSizeX, const uint& gridSizeY, const uint& numOutputClasses,
    const uint& numBBoxes, const void* anchors, hipStream_t stream)
{
  dim3 threads_per_block(16, 16, 4);
  dim3 number_of_blocks((gridSizeX / threads_per_block.x) + 1, (gridSizeY / threads_per_block.y) + 1,
      (numBBoxes / threads_per_block.z) + 1);

  for (unsigned int batch = 0; batch < batchSize; ++batch) {
    gpuRegionLayer<<<number_of_blocks, threads_per_block, 0, stream>>>(
        reinterpret_cast<const float*> (input) + (batch * inputSize),
        reinterpret_cast<float*> (softmax) + (batch * inputSize),
        reinterpret_cast<float*> (output) + (batch * 6 * outputSize),
        netWidth, netHeight, gridSizeX, gridSizeY, numOutputClasses, numBBoxes, lastInputSize,
        reinterpret_cast<const float*> (anchors));
  }
  return hipGetLastError();
}
